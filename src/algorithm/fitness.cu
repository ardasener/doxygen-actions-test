#include "hip/hip_runtime.h"
#ifndef INCLUDE_FITNESS_CU_
#define INCLUDE_FITNESS_CU_

#include "schedule.cpp"
#include <vector>
#include <algorithm>

#define CUDA_BLOCK_SIZE 256
#define MAX_ACADEMIC_BLOCKS 10
#define MAX_INSTS 5

//Penalties used to calculate fitness
struct Constants {
	int CUDA_SAME_DAY_PENALTY;
	int CUDA_INSTRUCTOR_COLLISION_PENALTY;
	int CUDA_CAPACITY_PENALTY;
	int CUDA_CLASSROOM_COLLISION_PENALTY;
	int CUDA_CONCURRENCY_COLLISION_PENALTY_MAX;
	int CUDA_ACADEMIC_BLOCK_COLLISION_PENALTY;
	int CUDA_INSTRUCTOR_TOO_MANY_CLASSES_PENALTY_MAX;
	int CUDA_INSTRUCTOR_TOO_MANY_CLASSES_PENALTY_MIN;
	int CUDA_CONSECUTIVE_DAYS_PENALTY;
	int CUDA_MAX_CONCURRENCY;
};

struct CudaCourseTime {
	int course_id;
	int group;
	int timeslot;
	int day;
	int room_capacity;
	int room_id;
	int course_capacity;
	int academic_blocks[MAX_ACADEMIC_BLOCKS];
	int instructors[MAX_INSTS];
	int academic_blocks_size;
	int instructors_size;
};



__global__ void calculateFitnessBulkCuda(CudaCourseTime* course_times, int* concurrency_matrix, int* result,
		int courses_size, int combined_size,
		int schedule_size, int number_of_schedules, Constants cons) {


	//Calculate indices

	int index = (blockIdx.x * blockDim.x) + threadIdx.x;

	if(index < 0 || index >= combined_size){
		return;
	}

	int schedule_index = index / schedule_size;

	if(schedule_index < 0 || schedule_index >= number_of_schedules){
		return;
	}

	int offset = schedule_index*schedule_size;


	//Calculate fitnesses
	unsigned int fitness = 0;


	CudaCourseTime* this_ct = &(course_times[index]);

	//Capacity check
	if(this_ct->room_capacity < this_ct->course_capacity){
		fitness += cons.CUDA_CAPACITY_PENALTY;
	}

	//All classes in same day check
	for(int i=index+1; i<offset+schedule_size && i<combined_size; i++){
		CudaCourseTime* other_ct = &(course_times[i]);

		if(other_ct->day != this_ct->day){
			break;
		}

		if(other_ct->course_id == this_ct->course_id && other_ct->group != this_ct->group){
			fitness += cons.CUDA_SAME_DAY_PENALTY;
		}

	}


	//Collision checks
	for(int i=index+1; i<offset + schedule_size && i < combined_size; i++){

		CudaCourseTime* other_ct = &(course_times[i]);

		if(this_ct->timeslot != other_ct->timeslot){
			//Since course times are sorted before this funtion begins
			//after this point all other coursetimes have non-colliding timeslots with this one.
			break;
		}

		//Room collision check
		if(this_ct->room_id == other_ct->room_id){
			fitness += cons.CUDA_CLASSROOM_COLLISION_PENALTY;
		}


		//Academic block collision check
		bool found = false;
		for(int j=0; !found && j < this_ct->academic_blocks_size && j < MAX_ACADEMIC_BLOCKS; j++){
			for(int k=0; !found && k< other_ct->academic_blocks_size && k < MAX_ACADEMIC_BLOCKS; k++){
				if(this_ct->academic_blocks[j] == other_ct->academic_blocks[k]){
					fitness += cons.CUDA_ACADEMIC_BLOCK_COLLISION_PENALTY;
					found = true;
				}
			}
		}


		//Instructor collision check
		found = false;
		for(int j=0; !found && j < this_ct->instructors_size && j < MAX_INSTS; j++){
			for(int k=0; !found &&  k< other_ct->instructors_size && k < MAX_INSTS; k++){
				if(this_ct->instructors[j] == other_ct->instructors[k]){
					fitness += cons.CUDA_INSTRUCTOR_COLLISION_PENALTY;
					found =  true;
				}
			}
		}

		//Concurrency collision check
		int concurrency_index = (this_ct->course_id * courses_size) + other_ct->course_id;
		int concurrent_penalty =  cons.CUDA_CONCURRENCY_COLLISION_PENALTY_MAX * ((float) concurrency_matrix[concurrency_index] / cons.CUDA_MAX_CONCURRENCY);
		if(concurrent_penalty > cons.CUDA_CONCURRENCY_COLLISION_PENALTY_MAX){
			concurrent_penalty = cons.CUDA_CONCURRENCY_COLLISION_PENALTY_MAX;
		}
		fitness += concurrent_penalty;


	}

	//Atomically write the results

	atomicAdd(&(result[schedule_index]), fitness);

}

__host__ void calculateFitnessBulk(std::vector<Schedule*>* schedules, int* concurrency_matrix_host, int courses_size, Config &config){

	hipSetDevice(config.device_num);

	Constants cons;

	cons.CUDA_SAME_DAY_PENALTY = config.SAME_DAY_PENALTY;
	cons.CUDA_INSTRUCTOR_COLLISION_PENALTY = config.INSTRUCTOR_COLLISION_PENALTY;
	cons.CUDA_CAPACITY_PENALTY = config.CAPACITY_PENALTY;
	cons.CUDA_CLASSROOM_COLLISION_PENALTY = config.CLASSROOM_COLLISION_PENALTY;
	cons.CUDA_CONCURRENCY_COLLISION_PENALTY_MAX = config.CONCURRENCY_COLLISION_PENALTY_MAX;
	cons.CUDA_ACADEMIC_BLOCK_COLLISION_PENALTY = config.ACADEMIC_BLOCK_COLLISION_PENALTY;
	cons.CUDA_INSTRUCTOR_TOO_MANY_CLASSES_PENALTY_MAX = config.INSTRUCTOR_TOO_MANY_CLASSES_PENALTY_MAX;
	cons.CUDA_INSTRUCTOR_TOO_MANY_CLASSES_PENALTY_MIN = config.INSTRUCTOR_TOO_MANY_CLASSES_PENALTY_MIN;
	cons.CUDA_CONSECUTIVE_DAYS_PENALTY = config.CONSECUTIVE_DAYS_PENALTY;
	cons.CUDA_MAX_CONCURRENCY = config.MAX_CONCURRENCY;





	//SORT THE SCHEDULES (NECESSARY)
	//TODO: Try to perform this on the gpu also
	for(auto pos = schedules->begin(); pos != schedules->end(); pos++){
		Schedule * schedule = (*pos);
		std::sort(schedule->getCourseTimes()->begin(), schedule->getCourseTimes()->end(), CourseTime::compareByTime);
	}

	//CREATE HOST VARIABLES

	int schedule_size = schedules->at(0)->getSize();
	//schedule_size = 2000;

	int number_of_schedules = schedules->size();

	int combined_array_size = number_of_schedules*schedule_size;

	int* result_array_host = new int[number_of_schedules];

	CudaCourseTime* course_times_host = new CudaCourseTime[combined_array_size];

	for(int i=0; i< number_of_schedules; i++){
		for(int j=0; j<schedule_size; j++){
			int index = (i*schedule_size) + j;
			CourseTime * ct = &(schedules->at(i)->getCourseTimes()->at(j));

			//std::cout << "New Course Time" << std::endl;

			CudaCourseTime cuda_ct;

			cuda_ct.course_id = ct->m_course->getId();
			cuda_ct.timeslot = ct->m_timeslot->m_id;
			cuda_ct.room_capacity = ct->m_classroom->m_capacity;
			cuda_ct.room_id = ct->m_classroom->m_id;
			cuda_ct.course_capacity = ct->m_course->getCapacity();
			cuda_ct.academic_blocks_size = ct->m_course->getBlocks()->size();
			cuda_ct.instructors_size = ct->m_course->getInstructorIds()->size();
			cuda_ct.group = ct->getGroup();
			cuda_ct.day = ct->m_timeslot->m_day;

			int k = 0;
			for(auto pos = ct->m_course->getBlocks()->begin(); pos != ct->m_course->getBlocks()->end(); pos++){
				if(k >= MAX_ACADEMIC_BLOCKS){
					break;
				}
				cuda_ct.academic_blocks[k] = *pos;
				//std::cout << "Academic Block: " << *pos << std::endl;
			}

			k = 0;
			for(auto pos = ct->m_course->getInstructorIds()->begin(); pos != ct->m_course->getInstructorIds()->end(); pos++){
				if(k >= MAX_INSTS){
					break;
				}
				cuda_ct.instructors[k] = *pos;
			}

			course_times_host[index] = cuda_ct;

		}
	}


	//CALCULATE THREAD / BLOCK SIZE
	int block_size = CUDA_BLOCK_SIZE;
	int num_blocks = (combined_array_size / block_size) + 1;

	/*
		 std::cout << "Number of schedules: " << number_of_schedules << ", Schedule Size: " << schedule_size << ", Combined Size: " << combined_array_size << std::endl;
		 std::cout << "Block Size: " << block_size << ", Num. Blocks:" << num_blocks << std::endl;
	 */

	//CREATE DEVICE VARIABLES

	CudaCourseTime* course_times_device;
	int* concurrency_matrix_device;
	int* result_array_device;



	hipMalloc((void **) &concurrency_matrix_device, sizeof(int)*courses_size*courses_size);
	hipMalloc((void **) &course_times_device, sizeof(CudaCourseTime)*combined_array_size);
	hipMalloc((void **) &result_array_device, sizeof(int)*number_of_schedules);


	//SET DEVICE VARIABLES

	hipMemset(result_array_device,0,sizeof(int)*number_of_schedules);


	//COPY HOST VARIABLES TO DEVICE

	hipMemcpy(course_times_device,course_times_host,sizeof(CudaCourseTime)*combined_array_size, hipMemcpyHostToDevice);
	hipMemcpy(concurrency_matrix_device, concurrency_matrix_host, sizeof(int)*courses_size*courses_size, hipMemcpyHostToDevice);

	//RUN THE KERNEL
	calculateFitnessBulkCuda<<<num_blocks,block_size>>>(course_times_device, concurrency_matrix_device , result_array_device, courses_size, combined_array_size, schedule_size, number_of_schedules,cons);


	//COPY DEVICE VARIABLES TO HOST
	hipDeviceSynchronize();
	hipMemcpy(result_array_host, result_array_device, sizeof(int)*number_of_schedules, hipMemcpyDeviceToHost);
	hipDeviceSynchronize();

	//std::cout << "Cuda Fitnesses: ";
	for(int i=0; i<number_of_schedules; i++){
		schedules->at(i)->setFitness(result_array_host[i]);
		//std::cout << result_array_host[i] << ", ";
	}
	//std::cout << std::endl;



	//DELETE (FREE) HOST MEMORY

	delete[] course_times_host;
	delete[] result_array_host;


	//DELETE (FREE) DEVICE MEMORY

	hipFree(course_times_device);
	hipFree(result_array_device);
	hipFree(concurrency_matrix_device);

}

#endif
