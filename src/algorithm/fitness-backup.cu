#include "hip/hip_runtime.h"
#ifndef INCLUDE_FITNESS_CU_
#define INCLUDE_FITNESS_CU_

#include "schedule.cpp"
#include <vector>


//Penalties used to calculate fitness
__constant__ __device__ int CUDA_INSTRUCTOR_COLLISION_PENALTY = 1000;
__constant__ __device__ int CUDA_CAPACITY_PENALTY = 900;
__constant__ __device__ int CUDA_CLASSROOM_COLLISION_PENALTY = 850;
__constant__ __device__ int CUDA_CONCURRENCY_COLLISION_PENALTY_MAX = 700;
__constant__ __device__ int CUDA_ACADEMIC_BLOCK_COLLISION_PENALTY = 500;
__constant__ __device__ int CUDA_INSTRUCTOR_TOO_MANY_CLASSES_PENALTY_MAX = 600;
__constant__ __device__ int CUDA_INSTRUCTOR_TOO_MANY_CLASSES_PENALTY_MIN = 200;
__constant__ __device__ int CUDA_CONSECUTIVE_DAYS_PENALTY = 200;

/*

class CudaConcurrent {
  
};

class CudaTimeSlot {
public:
  int m_day;
  int m_hour;
  int m_id;

  CudaTimeSlot(){
	
  }

 __host__ CudaTimeSlot(TimeSlot* t_t){
	m_day = t_t->m_day;
	m_hour = t_t->m_hour;
	m_id = t_t->m_id;
  }
};

class CudaClassroom {
public:
  int m_capacity;

  CudaClassroom(){
	
  }

 __host__ CudaClassroom(Classroom *t_c){
	m_capacity = t_c->m_capacity;
  }
};

class CudaCourse {
public:
  int m_capacity;

  CudaCourse(){
	
  }

 __host__ CudaCourse(Course *t_c){
	m_capacity = t_c->getCapacity();
  }
};

class CudaCourseTime {
public:
  CudaCourse m_course; 
  CudaClassroom m_classroom;
  CudaTimeSlot m_timeslot;

  CudaCourseTime(){
	
  }

  __host__ CudaCourseTime(CourseTime *t_ct){
	m_course = *(new CudaCourse(t_ct->m_course));
	m_classroom = *(new CudaClassroom(t_ct->m_classroom));
	m_timeslot = *(new CudaTimeSlot(t_ct->m_timeslot));
  }
};


class CudaSchedule {
public:
  CudaCourseTime * m_coursetimes;
  int m_size;

  CudaSchedule(){
	
  }

  __host__ CudaSchedule(std::vector<CourseTime> * t_coursetimes){
	m_size = t_coursetimes->size();
	//std::cout << "Size:" << m_size << std::endl;
	CudaCourseTime * coursetimes = new CudaCourseTime[t_coursetimes->size()];
	hipMalloc((void**) m_coursetimes, m_size*sizeof(CourseTime));

	for(int i=0; i < t_coursetimes->size(); i++){
	  coursetimes[0] = *(new CudaCourseTime(&(t_coursetimes->at(i))));
	}
	
	hipMemcpy(m_coursetimes, coursetimes, m_size*sizeof(CourseTime), hipMemcpyHostToDevice);  	
  }

  int getSize(){
	return m_size;
  }
};
*/

__global__ void calculateFitnessBulkCuda(CudaSchedule* schedules, int* result, int size) {

  int index = blockIdx.x * blockDim.x + threadIdx.x;

  if(index < 0 || index >= size){
	return;
  }

  int fitness = 0;

  CudaSchedule * schedule = &schedules[index];
  int schedule_size = schedule->m_size;

  for(int i=0; i<schedule_size; i++){
	CudaCourseTime * ct = &(schedule->m_coursetimes[i]);

	if(ct->m_course.m_capacity > ct->m_classroom.m_capacity){
	  fitness += CUDA_CAPACITY_PENALTY; 
	}
  }


  result[index] = schedule->m_coursetimes[0].m_course.m_capacity;
}


__host__ std::vector<int>* calculateFitnessBulk(std::vector<Schedule*>* schedules){

  int size = schedules->size();
  
  int* result_array;
  CudaSchedule* schedule_array = new CudaSchedule[size];

  for(int i=0; i<size; i++){
	std::vector<CourseTime> * coursetimes = schedules->at(i)->getCourseTimes();

	schedule_array[i] = *(new CudaSchedule(coursetimes));	
  }

  CudaSchedule* device_array;
  
  hipMalloc((void**)&device_array, size*sizeof(CudaSchedule));
  hipMalloc((void**)&result_array, size*sizeof(int));
  hipMemcpy(device_array, schedule_array, size*sizeof(CudaSchedule), hipMemcpyHostToDevice);  
  hipMemset(result_array, 0, size*sizeof(int));
  
  calculateFitnessBulkCuda<<<1,256>>>(device_array, result_array, size);


  int * host_result = new int[size];
  hipMemcpy(host_result, result_array, size*sizeof(int), hipMemcpyDeviceToHost);
  std::vector<int> * result_vector = new std::vector<int>(host_result,host_result + size - 1);
	
  return result_vector;
}

#endif